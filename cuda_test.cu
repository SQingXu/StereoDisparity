#include "hip/hip_runtime.h"
#include "cuda_test.h"
using namespace std;
using namespace cv;

char filename[] = "im2.png";
char dir[] = "/playpen/teddy";
char path[30];
int filter_size = 19;
int sigma = 3;
__global__
void blur_filter(uchar* orig, uchar* blur, int* fsize){
    int rows = gridDim.x;
    int cols = blockDim.x;
    int r = *fsize/2;
    int current_pos = blockIdx.x * cols + threadIdx.x;
    if((int)blockIdx.x - r < 0 || blockIdx.x + r >= rows || (int)threadIdx.x - r < 0 || threadIdx.x + r >= cols){
        blur[current_pos] = 0;
    }else{
        int total = 0;
        for(int i = -r; i < r+1; i++){
            for(int j = -r; j < r+1; j++){
                total += (int)orig[current_pos + i*cols + j];
            }
        }
        blur[current_pos] = (uchar)(total/(float)(*fsize * *fsize));
    }
}

__global__
void gaussian_filter(uchar* orig, uchar* g_res, int* sigma, int* fsize){
    int rows = gridDim.x;
    int cols = blockDim.x;
    int r = *fsize/2;
    int cpos = blockIdx.x * cols + threadIdx.x;
    if((int)blockIdx.x - r < 0 || blockIdx.x + r >= rows || (int)threadIdx.x - r < 0 || threadIdx.x + r >= cols){
        g_res[cpos] = 0;
    }else{
        int g_val = 0;
        for(int i = -r; i < r+1 ; i++){
            for(int j = -r; j < r+1; j++){
                float gc = (1/(2*3.1415926*(*sigma)*(*sigma)))*expf(-1*(i*i+j*j)/((float)2*(*sigma)*(*sigma)));
                g_val += (gc * orig[cpos + i*cols + j]);
            }
        }
        g_res[cpos] = (uchar)g_val;
    }
}

__global__
void edge_detector(uchar* orig, uchar* edge){
    int rows = gridDim.x;
    int cols = blockDim.x;
    int cpos = blockIdx.x * cols + threadIdx.x;
    if(blockIdx.x == 0 || threadIdx.x == 0 || blockIdx.x ==rows-1 || threadIdx.x == cols-1){
        edge[cpos] = 0;
    }else{
        //apply sobel filter
        int totalx = 0, totaly = 0;
        int frows = -1,fcols = -1;
        for(frows = -1; frows < 2; frows++){
            for(fcols = -1; fcols < 2; fcols++){
                int xsign = (fcols == -1)?1:(fcols == 0)?0:-1;
                int xc = (frows == 0)?2:1;
                int ysign = (frows == -1)?1:(frows == 0)?0:-1;
                int yc = (fcols == 0)?2:1;

                totalx += (int)orig[cpos + frows * cols + fcols] * (xc * xsign);
                totaly += (int)orig[cpos + frows * cols + fcols] * (yc * ysign);
            }
        }
        edge[cpos] = (uchar)sqrtf(totalx * totalx + totaly * totaly);
    }
}

//void add_v(int *a, int *b, int *c){
//    //*c = *a + *b;
//    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
//}

extern __device__ int shared_img[256];

void random_ints(int *a, int range){
    int i;
    for(i = 0; i < range; i++){
        a[i] = rand()%50;
    }
}

void print_vector(int* a){
    int i  = 0;
    for(i = 0; i < N; i++){
        if(i == N -1){
            printf("%d\n", a[i]);
            continue;
        }
        printf("%d ",a[i]);
    }
}

//int main(void){
////    int *a,*b;
////    int *c;
////    int *d_a, *d_b, *d_c;
////    int size = N * sizeof(int);

////    hipMalloc((void **) &d_a, size);
////    hipMalloc((void **) &d_b, size);
////    hipMalloc((void **) &d_c, size);
//    uchar* d_orig;
//    uchar* d_res, *res, * d_gres;
//    int* d_fsize;
//    int* d_sigma;

//    sprintf(path, "%s/%s", dir, filename);
//    Mat img = imread(path, CV_LOAD_IMAGE_GRAYSCALE);
//    if(!img.data){
//        return 1;
//    }
//    int rows = img.rows;
//    int cols = img.cols;
//    int img_dim = cols * rows;
//    printf("size is %d\n", img_dim);

//    int size = img_dim * sizeof(uchar);
//    //initialization
//    hipMalloc((void **) &d_sigma, sizeof(int));
//    hipMalloc((void **) &d_fsize, sizeof(int));
//    hipMalloc((void **) &d_orig, size);
//    hipMalloc((void **) &d_res, size);
//    hipMalloc((void **) &d_gres, size);
//    res = (uchar*)malloc(size);

//    DFS_CUDA_ASSERT(hipMemcpy(d_fsize, &filter_size, sizeof(int), hipMemcpyHostToDevice));
//    DFS_CUDA_ASSERT(hipMemcpy(d_sigma, &sigma,sizeof(int),hipMemcpyHostToDevice));
//    DFS_CUDA_ASSERT(hipMemcpy(d_orig, img.data, size, hipMemcpyHostToDevice));
//    //blur_filter<<<rows, cols>>>(d_orig, d_res, d_fsize);

//    gaussian_filter<<<rows,cols>>>(d_orig,d_gres,d_sigma,d_fsize);
//    edge_detector<<<rows, cols>>>(d_gres,d_res);

//    DFS_CUDA_ASSERT(hipPeekAtLastError());
//    DFS_CUDA_ASSERT(hipMemcpy(res, d_res, size, hipMemcpyDeviceToHost));

//    DFS_CUDA_ASSERT(hipFree(d_fsize));
//    DFS_CUDA_ASSERT(hipFree(d_sigma));
//    DFS_CUDA_ASSERT(hipFree(d_gres));
//    DFS_CUDA_ASSERT(hipFree(d_orig));
//    DFS_CUDA_ASSERT(hipFree(d_res));


//    Mat output(rows,cols,CV_8UC1,res);
//    imwrite("Guassian3&Edges.png", output);
//    namedWindow("Original", WINDOW_AUTOSIZE );
//    imshow("Original",img);
//    namedWindow("Gaussian", WINDOW_AUTOSIZE );
//    imshow("Gaussian",output);
//    waitKey(0);
//    free(res);



////    a = (int *)malloc(size);
////    b = (int *)malloc(size);
////    random_ints(a,N);
////    random_ints(b,N);
////    c = (int *)malloc(size);

////    DFS_CUDA_ASSERT(hipMemcpy(d_a, a, size, hipMemcpyHostToDevice));
////    DFS_CUDA_ASSERT(hipMemcpy(d_b, b, size, hipMemcpyHostToDevice));
////    add_v<<<1,N>>>(d_a, d_b, d_c);
////    DFS_CUDA_ASSERT(hipPeekAtLastError());
////    DFS_CUDA_ASSERT(hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost));

////    print_vector(a);
////    print_vector(b);
////    print_vector(c);

////    free(a);
////    free(b);
////    free(c);
////    DFS_CUDA_ASSERT(hipFree(d_a));
////    DFS_CUDA_ASSERT(hipFree(d_b));
////    DFS_CUDA_ASSERT(hipFree(d_c));

//    return 0;
//}
